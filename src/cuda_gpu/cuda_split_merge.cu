#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "../common/image_io.h"

#define VAR_THRESHOLD 500
#define DIFF_THRESHOLD 10

#define BLOCK_SIZE 16  // CUDA block size

// Kernel: Initialize labels (each pixel gets its own label)
__global__ void init_labels(uint8_t *img, int *labels, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        labels[y * width + x] = y * width + x;  // unique label
    }
}

// Kernel: Merge neighboring pixels based on intensity difference
__global__ void merge_labels(uint8_t *img, int *labels, int width, int height, int *changed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width - 1 && y < height - 1) {
        int idx = y * width + x;
        int right = idx + 1;
        int down = idx + width;

        if (abs(img[idx] - img[right]) < DIFF_THRESHOLD) {
            int min_label = min(labels[idx], labels[right]);
            if (labels[right] != min_label) {
                labels[right] = min_label;
                *changed = 1;
            }
            if (labels[idx] != min_label) {
                labels[idx] = min_label;
                *changed = 1;
            }
        }

        if (abs(img[idx] - img[down]) < DIFF_THRESHOLD) {
            int min_label = min(labels[idx], labels[down]);
            if (labels[down] != min_label) {
                labels[down] = min_label;
                *changed = 1;
            }
            if (labels[idx] != min_label) {
                labels[idx] = min_label;
                *changed = 1;
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s input.pgm output.pgm\n", argv[0]);
        return -1;
    }

    Image *img = read_pgm(argv[1]);
    int size = img->width * img->height;

    uint8_t *d_img;
    int *d_labels, *d_changed;
    int changed;

    hipMalloc(&d_img, size * sizeof(uint8_t));
    hipMalloc(&d_labels, size * sizeof(int));
    hipMalloc(&d_changed, sizeof(int));

    hipMemcpy(d_img, img->data, size * sizeof(uint8_t), hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((img->width + BLOCK_SIZE - 1) / BLOCK_SIZE, (img->height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    init_labels<<<grid, block>>>(d_img, d_labels, img->width, img->height);
    hipDeviceSynchronize();

    do {
        changed = 0;
        hipMemcpy(d_changed, &changed, sizeof(int), hipMemcpyHostToDevice);

        merge_labels<<<grid, block>>>(d_img, d_labels, img->width, img->height, d_changed);
        hipDeviceSynchronize();

        hipMemcpy(&changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
    } while (changed);

    int *labels = (int*)malloc(size * sizeof(int));
    hipMemcpy(labels, d_labels, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        img->data[i] = labels[i] % 256;
    }

    write_pgm(argv[2], img);

    free(labels);
    free_image(img);
    hipFree(d_img);
    hipFree(d_labels);
    hipFree(d_changed);
    return 0;
}
