#include <stdint.h>
#include <hip/hip_runtime.h>

#define DIFF_THRESHOLD 10
#define BLOCK_SIZE 16

__global__ void init_labels(uint8_t *img, int *labels, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        labels[y * width + x] = y * width + x;
    }
}

__global__ void merge_labels(uint8_t *img, int *labels, int width, int height, int *changed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width - 1 && y < height - 1) {
        int idx = y * width + x;
        int right = idx + 1;
        int down = idx + width;
        if (abs(img[idx] - img[right]) < DIFF_THRESHOLD) {
            int min_label = min(labels[idx], labels[right]);
            if (labels[right] != min_label) {
                labels[right] = min_label;
                *changed = 1;
            }
            if (labels[idx] != min_label) {
                labels[idx] = min_label;
                *changed = 1;
            }
        }
        if (abs(img[idx] - img[down]) < DIFF_THRESHOLD) {
            int min_label = min(labels[idx], labels[down]);
            if (labels[down] != min_label) {
                labels[down] = min_label;
                *changed = 1;
            }
            if (labels[idx] != min_label) {
                labels[idx] = min_label;
                *changed = 1;
            }
        }
    }
}


extern "C" void cuda_init_labels(uint8_t **d_img, int **d_labels, int **d_changed, uint8_t *local_data, int width, int height_per_proc)
{

    hipMalloc(d_img, width * height_per_proc * sizeof(uint8_t));
    hipMalloc(d_labels, width * height_per_proc * sizeof(int));
    hipMalloc(d_changed, sizeof(int));

    hipMemcpy(*d_img, local_data, width * height_per_proc * sizeof(uint8_t), hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height_per_proc + BLOCK_SIZE - 1) / BLOCK_SIZE);

    init_labels<<<grid, block>>>(*d_img, *d_labels, width, height_per_proc);
    hipDeviceSynchronize();
}

extern "C" void cuda_merge_labels(int *d_changed, int *changed, uint8_t *d_img, int *d_labels, int *labels, int width, int height_per_proc)
{
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height_per_proc + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipMemcpy(labels, d_labels, width * height_per_proc * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(d_changed, changed, sizeof(int), hipMemcpyHostToDevice);

    merge_labels<<<grid, block>>>(d_img, d_labels, width, height_per_proc, d_changed);
    hipDeviceSynchronize();

    hipMemcpy(changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(labels, d_labels, width * height_per_proc * sizeof(int), hipMemcpyDeviceToHost);
}

extern "C" void cuda_update_labels(int *labels, int *d_labels, int width, int height_per_proc) {
    hipMemcpy(d_labels, labels, width * height_per_proc * sizeof(int), hipMemcpyHostToDevice);
}

extern "C" void cuda_free(uint8_t *d_img, int *d_labels, int *d_changed)
{
    hipFree(d_img);
    hipFree(d_labels);
    hipFree(d_changed);
}