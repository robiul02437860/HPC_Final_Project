#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

#include "../common/image_io.h"

#define DIFF_THRESHOLD 10
#define BLOCK_SIZE 16

__global__ void init_labels(uint8_t *img, int *labels, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        labels[y * width + x] = y * width + x;
    }
}

__global__ void merge_labels(uint8_t *img, int *labels, int width, int height, int *changed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width - 1 && y < height - 1) {
        int idx = y * width + x;
        int right = idx + 1;
        int down = idx + width;
        if (abs(img[idx] - img[right]) < DIFF_THRESHOLD) {
            int min_label = min(labels[idx], labels[right]);
            if (labels[right] != min_label) {
                labels[right] = min_label;
                *changed = 1;
            }
            if (labels[idx] != min_label) {
                labels[idx] = min_label;
                *changed = 1;
            }
        }
        if (abs(img[idx] - img[down]) < DIFF_THRESHOLD) {
            int min_label = min(labels[idx], labels[down]);
            if (labels[down] != min_label) {
                labels[down] = min_label;
                *changed = 1;
            }
            if (labels[idx] != min_label) {
                labels[idx] = min_label;
                *changed = 1;
            }
        }
    }
}

void exchange_boundaries(int *labels, int width, int height, int rank, int size, MPI_Comm comm) {
    MPI_Status status;
    if (rank != 0) {
        MPI_Sendrecv(labels, width, MPI_INT, rank - 1, 0,
                     labels - width, width, MPI_INT, rank - 1, 0, comm, &status);
    }
    if (rank != size - 1) {
        MPI_Sendrecv(labels + (height - 1) * width, width, MPI_INT, rank + 1, 0,
                     labels + height * width, width, MPI_INT, rank + 1, 0, comm, &status);
    }
}

int main(int argc, char *argv[]) {
    int rank, size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    if (argc != 3) {
        if (rank == 0)
            printf("Usage: %s input.pgm output.pgm\n", argv[0]);
        MPI_Finalize();
        return -1;
    }

    Image *img = NULL;
    if (rank == 0)
        img = read_pgm(argv[1]);

    int width, total_height;
    if (rank == 0) {
        width = img->width;
        total_height = img->height;
    }

    MPI_Bcast(&width, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&total_height, 1, MPI_INT, 0, MPI_COMM_WORLD);

    int height_per_proc = total_height / size;

    uint8_t *local_data = (uint8_t*)malloc(width * height_per_proc * sizeof(uint8_t));
    MPI_Scatter(img ? img->data : NULL, width * height_per_proc, MPI_UINT8_T,
                local_data, width * height_per_proc, MPI_UINT8_T, 0, MPI_COMM_WORLD);

    uint8_t *d_img;
    int *d_labels, *d_changed;
    int changed;
    hipMalloc(&d_img, width * height_per_proc * sizeof(uint8_t));
    hipMalloc(&d_labels, width * height_per_proc * sizeof(int));
    hipMalloc(&d_changed, sizeof(int));

    hipMemcpy(d_img, local_data, width * height_per_proc * sizeof(uint8_t), hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height_per_proc + BLOCK_SIZE - 1) / BLOCK_SIZE);

    init_labels<<<grid, block>>>(d_img, d_labels, width, height_per_proc);
    hipDeviceSynchronize();

    do {
        changed = 0;
        hipMemcpy(d_changed, &changed, sizeof(int), hipMemcpyHostToDevice);

        merge_labels<<<grid, block>>>(d_img, d_labels, width, height_per_proc, d_changed);
        hipDeviceSynchronize();

        hipMemcpy(&changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
        MPI_Allreduce(MPI_IN_PLACE, &changed, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD);

        exchange_boundaries(d_labels, width, height_per_proc, rank, size, MPI_COMM_WORLD);

    } while (changed);

    int *labels = (int*)malloc(width * height_per_proc * sizeof(int));
    hipMemcpy(labels, d_labels, width * height_per_proc * sizeof(int), hipMemcpyDeviceToHost);

    uint8_t *output_data = (uint8_t*)malloc(width * height_per_proc * sizeof(uint8_t));
    for (int i = 0; i < width * height_per_proc; i++) {
        output_data[i] = labels[i] % 256;
    }

    if (rank == 0) {
        uint8_t *full_output = (uint8_t*)malloc(width * total_height * sizeof(uint8_t));
        MPI_Gather(output_data, width * height_per_proc, MPI_UINT8_T,
                   full_output, width * height_per_proc, MPI_UINT8_T, 0, MPI_COMM_WORLD);
        img->data = full_output;
        write_pgm(argv[2], img);
        free_image(img);
    } else {
        MPI_Gather(output_data, width * height_per_proc, MPI_UINT8_T,
                   NULL, 0, MPI_UINT8_T, 0, MPI_COMM_WORLD);
    }

    free(local_data);
    free(labels);
    free(output_data);
    hipFree(d_img);
    hipFree(d_labels);
    hipFree(d_changed);

    MPI_Finalize();
    return 0;
}
